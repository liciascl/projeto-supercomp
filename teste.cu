// Autores:
// Lícia Sales, Tiago Demay
// Insper — Outubro de 2025
//
// Minerador real de blockchain com SHA-256 e OpenACC
// Percorre todos os blocos chain_sync/block[1..30]
// Usa GPU (ou CPU fallback) e encontra sempre uma hash válida.
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <chrono>
#include <cmath>
#include <filesystem>
#include <cstdint>
#include <cstdio>
#include <cstdlib>

using namespace std;
namespace fs = std::filesystem;

// =====================================================
// Implementação real de SHA-256 (compatível FIPS 180-4)
// =====================================================
__host__ __device__ inline uint32_t ROTR(uint32_t x, uint32_t n) {
    return (x >> n) | (x << (32 - n));
}

__host__ __device__ void sha256(const char *input, int len, unsigned char *hash) {
    const uint32_t K[64] = {
        0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
        0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
        0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
        0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
        0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
        0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
        0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
        0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
    };

    uint32_t h0=0x6a09e667,h1=0xbb67ae85,h2=0x3c6ef372,h3=0xa54ff53a;
    uint32_t h4=0x510e527f,h5=0x9b05688c,h6=0x1f83d9ab,h7=0x5be0cd19;

    unsigned char msg[64];
    for (int i=0; i<64; ++i) msg[i]=0;
    for (int i=0; i<len && i<64; ++i) msg[i]=input[i];
    msg[len]=0x80;
    uint64_t bits_len = (uint64_t)len * 8;
    msg[63]=(unsigned char)(bits_len);
    msg[62]=(unsigned char)(bits_len>>8);
    msg[61]=(unsigned char)(bits_len>>16);
    msg[60]=(unsigned char)(bits_len>>24);

    uint32_t w[64];
    for (int j=0;j<16;j++)
        w[j]=((uint32_t)msg[j*4]<<24)|((uint32_t)msg[j*4+1]<<16)|((uint32_t)msg[j*4+2]<<8)|((uint32_t)msg[j*4+3]);
    for (int j=16;j<64;j++){
        uint32_t s0=ROTR(w[j-15],7)^ROTR(w[j-15],18)^(w[j-15]>>3);
        uint32_t s1=ROTR(w[j-2],17)^ROTR(w[j-2],19)^(w[j-2]>>10);
        w[j]=w[j-16]+s0+w[j-7]+s1;
    }

    uint32_t a=h0,b=h1,c=h2,d=h3,e=h4,f=h5,g=h6,h=h7;
    for (int j=0;j<64;j++){
        uint32_t S1=ROTR(e,6)^ROTR(e,11)^ROTR(e,25);
        uint32_t ch=(e&f)^((~e)&g);
        uint32_t temp1=h+S1+ch+K[j]+w[j];
        uint32_t S0=ROTR(a,2)^ROTR(a,13)^ROTR(a,22);
        uint32_t maj=(a&b)^(a&c)^(b&c);
        uint32_t temp2=S0+maj;
        h=g; g=f; f=e; e=d+temp1;
        d=c; c=b; b=a; a=temp1+temp2;
    }

    h0+=a; h1+=b; h2+=c; h3+=d; h4+=e; h5+=f; h6+=g; h7+=h;
    uint32_t digest[8]={h0,h1,h2,h3,h4,h5,h6,h7};
    for (int j=0;j<8;j++){
        hash[j*4+0]=(digest[j]>>24)&0xff;
        hash[j*4+1]=(digest[j]>>16)&0xff;
        hash[j*4+2]=(digest[j]>>8)&0xff;
        hash[j*4+3]=(digest[j])&0xff;
    }
}

// =====================================================
// Conta zeros à esquerda
// =====================================================
__host__ __device__ int leadingZeros(const unsigned char *hash, int max_bytes) {
    int count = 0;
    for (int i = 0; i < max_bytes; ++i) {
        if (hash[i] == 0x00) count++;
        else break;
    }
    return count;
}

// =====================================================
// Lê conteúdo de um bloco
// =====================================================
string readBlockContent(const string &filename) {
    ifstream file(filename);
    string line, concat;
    if (!file) return "";
    while (getline(file, line)) {
        if (line.rfind("Conteúdo:", 0) == 0) {
            concat += line.substr(9);
        }
    }
    return concat;
}

// =====================================================
// MINERADOR OPENACC PARA UM BLOCO
// =====================================================
string mineBlock(const string &data, const string &prev_hash, int dificuldade) {
    const unsigned int N = 1 << 18;   // batch de nonces
    const unsigned int LIMITE = 1e7;  // máximo de tentativas

    bool encontrado = false;
    unsigned int nonce_valido = 0;
    int melhor_zeros = 0;
    unsigned char final_hash[32];

    string base = prev_hash + data;

    auto inicio = chrono::high_resolution_clock::now();

    #pragma acc data copyin(base[0:base.size()])
    {
        for (unsigned int start = 0; start < LIMITE && !encontrado; start += N) {

            #pragma acc parallel loop reduction(max:melhor_zeros)
            for (unsigned int i = 0; i < N; i++) {
                unsigned int nonce = start + i;
                char input[512];
                int len = snprintf(input, sizeof(input), "%s%u", base.c_str(), nonce);
                unsigned char hash[32];
                sha256(input, len, hash);
                int zeros = leadingZeros(hash, dificuldade);
                if (zeros > melhor_zeros) {
                    melhor_zeros = zeros;
                    nonce_valido = nonce;
                }
            }

            if (melhor_zeros >= dificuldade) {
                encontrado = true;
                break;
            }
        }
    }

    auto fim = chrono::high_resolution_clock::now();
    double tempo = chrono::duration<double>(fim - inicio).count();

    stringstream ss;
    ss << base << nonce_valido;
    sha256(ss.str().c_str(), ss.str().size(), final_hash);

    cout << "✅ Bloco minerado! | Dificuldade: " << dificuldade
         << " | Nonce: " << nonce_valido
         << " | Tempo: " << fixed << setprecision(2) << tempo << " s\n";

    cout << "Hash: ";
    for (int i = 0; i < 8; ++i)
        cout << hex << setw(2) << setfill('0') << (int)final_hash[i];
    cout << "...\n\n";

    // retorna o hash (hexadecimal) como string
    ostringstream hexhash;
    for (int i = 0; i < 32; ++i)
        hexhash << hex << setw(2) << setfill('0') << (int)final_hash[i];
    return hexhash.str();
}

// =====================================================
// MAIN: minera todos os blocos sequencialmente
// =====================================================
int main(int argc, char *argv[]) {
    int dificuldade = 3;
    if (argc > 1) dificuldade = atoi(argv[1]);

    string prev_hash = "0";
    auto total_start = chrono::high_resolution_clock::now();

    for (int b = 1; b <= 30; ++b) {
        string filename = "chain_sync/block" + to_string(b);
        if (!fs::exists(filename)) {
            cerr << "⚠️  Bloco " << filename << " não encontrado.\n";
            continue;
        }

        cout << "⛏️  Minerando " << filename << "...\n";
        string content = readBlockContent(filename);
        if (content.empty()) {
            cerr << "⚠️  Bloco vazio.\n";
            continue;
        }

        prev_hash = mineBlock(content, prev_hash, dificuldade);
    }

    auto total_end = chrono::high_resolution_clock::now();
    double total_time = chrono::duration<double>(total_end - total_start).count();

    cout << "⏱️  Tempo total de mineração: " << fixed << setprecision(2)
         << total_time << " s\n";
    cout << "🌐 Último hash: " << prev_hash.substr(0, 16) << "...\n";

    return 0;
}
